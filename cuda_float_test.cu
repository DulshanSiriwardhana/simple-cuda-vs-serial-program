
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define N 10000
#define STEPS 100

__global__ void update(double *A, double *A_new) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) A_new[i] = A[i] * 1.0000001 + 0.0000001 * (i % 3 - 1);
}

int main() {
    double A[N], A_new[N];
    double *d_A, *d_A_new;
    for (int i = 0; i < N; i++) A[i] = i * i * 0.0001;
    hipMalloc(&d_A, N * sizeof(double));
    hipMalloc(&d_A_new, N * sizeof(double));
    hipMemcpy(d_A, A, N * sizeof(double), hipMemcpyHostToDevice);
    int threads = N, blocks = 1;
    for (int s = 0; s < STEPS; s++) {
        update<<<blocks, threads>>>(d_A, d_A_new);
        double *tmp = d_A; d_A = d_A_new; d_A_new = tmp;
    }
    hipMemcpy(A, d_A, N * sizeof(double), hipMemcpyDeviceToHost);
    FILE *f = fopen("cuda_output.txt", "w");
    for (int i = 0; i < N; i++) fprintf(f, "%d %.20f\n", i, A[i]);
    fclose(f);
    hipFree(d_A); hipFree(d_A_new);
    return 0;
}