
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define N 10
#define STEPS 100

__global__ void update(double *A, double *A_new) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i > 0 && i < N - 1) {
        A_new[i] = A[i] + 0.1 * (A[i - 1] - 2 * A[i] + A[i + 1]);
    }
}

int main() {
    double *A, *A_new;
    double *d_A, *d_A_new;
    A = (double*)malloc(N * sizeof(double));
    A_new = (double*)malloc(N * sizeof(double));
    for (int i = 0; i < N; i++) {
        A[i] = i * i * 0.0001;
    }
    hipMalloc(&d_A, N * sizeof(double));
    hipMalloc(&d_A_new, N * sizeof(double));
    hipMemcpy(d_A, A, N * sizeof(double), hipMemcpyHostToDevice);
    int threads = 256;
    int blocks = (N + threads - 1) / threads;
    for (int step = 0; step < STEPS; step++) {
        hipDeviceSynchronize();
        update<<<blocks, threads>>>(d_A, d_A_new);
        hipDeviceSynchronize();
        double *temp = d_A;
        d_A = d_A_new;
        d_A_new = temp;
        hipDeviceSynchronize();
    }
    hipMemcpy(A, d_A, N * sizeof(double), hipMemcpyDeviceToHost);
    FILE *fp = fopen("cuda_output.txt", "w");
    if (fp == NULL) {
        fprintf(stderr, "Failed to open output file.\n");
        return 1;
    }
    for (int i = 0; i < N; i++) {
        fprintf(fp, "%d %.20f\n", i, A[i]);
    }
    fclose(fp);
    hipFree(d_A);
    hipFree(d_A_new);
    free(A);
    free(A_new);
    return 0;
} 